#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
//#include <ATen/native/cuda/DeviceSqrt.cuh>

#include <THC/THC.h>
#include <THC/THCDeviceUtils.cuh>

#include <vector>
#include <iostream>

int const threadsPerBlock = sizeof(unsigned long long) * 8;

__device__ inline float devIoU(float const * const a, float const * const b) {
  float left = max(a[0], b[0]), right = min(a[2], b[2]);
  float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
  float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return interS / (Sa + Sb - interS);
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 5];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * 5;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * 5) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = THCCeilDiv(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

__global__ void center_nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 7];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 7 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 0];
    block_boxes[threadIdx.x * 7 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 1];
    block_boxes[threadIdx.x * 7 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 2];
    block_boxes[threadIdx.x * 7 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 3];
    block_boxes[threadIdx.x * 7 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 4];
    block_boxes[threadIdx.x * 7 + 5] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 5];
    block_boxes[threadIdx.x * 7 + 6] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 6];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * 7;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      float ibx = (cur_box[0] + cur_box[2]) / 2;
      float iby = (cur_box[1] + cur_box[3]) / 2;
      float jbx = ((block_boxes + i * 7)[0] + (block_boxes + i * 7)[2]) / 2;
      float jby = ((block_boxes + i * 7)[1] + (block_boxes + i * 7)[3]) / 2;
      float icx = cur_box[5];
      float icy = cur_box[6];
      float jcx = (block_boxes + i * 7)[5];
      float jcy = (block_boxes + i * 7)[6];
      float weight = sqrtf((ibx - jbx) * (ibx - jbx) + (iby - jby) * (iby - jby)) / sqrtf((icx - jcx) * (icx - jcx) + (icy - jcy) * (icy - jcy));
      if (devIoU(cur_box, block_boxes + i * 7) * weight > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = THCCeilDiv(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

// boxes is a N x 5 tensor
at::Tensor nms_cuda(const at::Tensor boxes, float nms_overlap_thresh) {
  using scalar_t = float;
  AT_ASSERTM(boxes.type().is_cuda(), "boxes must be a CUDA tensor");
  auto scores = boxes.select(1, 4);
  auto order_t = std::get<1>(scores.sort(0, /* descending=*/true));
  auto boxes_sorted = boxes.index_select(0, order_t);

  int boxes_num = boxes.size(0);
  int input_cols = boxes.size(1);

  const int col_blocks = THCCeilDiv(boxes_num, threadsPerBlock);

  scalar_t* boxes_dev = boxes_sorted.data<scalar_t>();

  THCState *state = at::globalContext().lazyInitCUDA(); // TODO replace with getTHCState

  unsigned long long* mask_dev = NULL;
  //THCudaCheck(THCudaMalloc(state, (void**) &mask_dev,
  //                      boxes_num * col_blocks * sizeof(unsigned long long)));

  mask_dev = (unsigned long long*) THCudaMalloc(state, boxes_num * col_blocks * sizeof(unsigned long long));

  dim3 blocks(THCCeilDiv(boxes_num, threadsPerBlock),
              THCCeilDiv(boxes_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);
  if (input_cols == 7) {
    center_nms_kernel<<<blocks, threads>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  mask_dev);
  } else {
    nms_kernel<<<blocks, threads>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  mask_dev);

  }

  std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
  THCudaCheck(hipMemcpy(&mask_host[0],
                        mask_dev,
                        sizeof(unsigned long long) * boxes_num * col_blocks,
                        hipMemcpyDeviceToHost));

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  at::Tensor keep = at::empty({boxes_num}, boxes.options().dtype(at::kLong).device(at::kCPU));
  int64_t* keep_out = keep.data<int64_t>();

  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i;
      unsigned long long *p = &mask_host[0] + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }

  THCudaFree(state, mask_dev);
  // TODO improve this part
  return std::get<0>(order_t.index({
                       keep.narrow(/*dim=*/0, /*start=*/0, /*length=*/num_to_keep).to(
                         order_t.device(), keep.scalar_type())
                     }).sort(0, false));
}
